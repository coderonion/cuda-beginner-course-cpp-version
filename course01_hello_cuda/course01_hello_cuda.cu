
#include <hip/hip_runtime.h>
#include <cstdio>

// CUDA核函数（CPU主机端调用，GPU设备端执行）
// CUDA核函数标识符：__global__
__global__ void hello_cuda_from_gpu() {
    printf("GPU: 你好, CUDA! (C++版)\n");
}

// 普通函数5（CPU主机端调用和执行）
void hello_cuda_from_cpu() {
    printf("CPU: 你好, CUDA! (C++版)\n");
}

int main() {
    // GPU: 你好, CUDA! (C++版)
    {
        const int GRID_DIM = 2;          // Grid(网格)大小（即1个Grid中的线程块数量）
        const int BLOCK_DIM = 8;         // Block(线程块)大小（即1个Block中的线程数量）
        // CUDA核函数调用, 核函数配置参数<<<...>>>, 核函数总线程数为2*8=16
        hello_cuda_from_gpu<<<GRID_DIM, BLOCK_DIM>>>();
        hipDeviceSynchronize();        // 同步CPU主机端和GPU设备端
    }
    printf("\n");
    // CPU: 你好, CUDA! (C++版)
    {
        for(int i = 0; i < 2 * 8; ++i) {
            hello_cuda_from_cpu();
        }
    }
    return 0;
}