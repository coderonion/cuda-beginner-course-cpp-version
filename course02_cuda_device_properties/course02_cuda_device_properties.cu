#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>

using namespace std;

int main()
{
    // CUDA device count
    int cuda_device_count = 0;
    hipError_t cuda_error_info = hipGetDeviceCount(&cuda_device_count);
    if(cuda_error_info != hipSuccess) {
        std::cout << "hipGetDeviceCount error info: " << cuda_error_info << std::endl;
        return -1;
    } else {
        printf("Detected %d CUDA Capable device(s)\n", cuda_device_count);
    }
    for(int device_id = 0; device_id < cuda_device_count; ++device_id) {
        cuda_error_info = hipSetDevice(device_id);
        if(cuda_error_info != hipSuccess) {
            std::cout << "hipSetDevice error info: " << cuda_error_info << std::endl;
            continue;
        }
        // CUDA device properties
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device_id);
        // CUDA device name
        char* device_name = prop.name;
        // CUDA device compute capability
        int device_major_compute_capability = prop.major;
        int device_minor_compute_capability = prop.minor;
        // CUDA Device Global memory available on device in bytes
        size_t device_total_global_mem = prop.totalGlobalMem;
        // Clock frequency in kilohertz
        int device_clock_rate = prop.clockRate;
        // Peak memory clock frequency in kilohertz
        int device_memory_clock_rate = prop.memoryClockRate;
        // Global memory bus width in bits
        int device_memory_bus_width = prop.memoryBusWidth;
        // Size of L2 cache in bytes
        int device_l2_cache_size = prop.l2CacheSize;
        // CUDA Device Constant memory available on device in bytes
        size_t device_total_const_mem = prop.totalConstMem;
        // Shared memory available per block in bytes
        size_t device_shared_mem_per_block = prop.sharedMemPerBlock;
        // 32-bit registers available per block
        int device_regs_per_block = prop.regsPerBlock;
        // Warp size in threads
        int device_warp_size = prop.warpSize;
        // Maximum resident threads per multiprocessor
        int device_max_threads_per_multi_processor = prop.maxThreadsPerMultiProcessor;
        // Maximum number of threads per block
        int device_max_threads_per_block = prop.maxThreadsPerBlock;
        // Maximum size of each dimension of a block
        int* device_max_threads_dim = prop.maxThreadsDim;
        // Maximum size of each dimension of a grid
        int* device_max_grid_size = prop.maxGridSize;
        printf(" Device %d: %s\n", device_id, device_name);
        printf(" CUDA Capability Major/Minor version number:    %d.%d\n", device_major_compute_capability, device_minor_compute_capability);
        printf(" Total amount of global memory:                 %.0lf MBytes (%lld bytes)\n", device_total_global_mem / (1024.0 * 1024.0), device_total_global_mem);
        printf(" GPU Max Clock rate:                            %.0f MHz\n", device_clock_rate / (1000.0));
        printf(" Memory Clock rate:                             %.0f MHz\n", device_memory_clock_rate / (1000.0));
        printf(" Memory Bus Width:                              %d-bit\n", device_memory_bus_width);
        printf(" L2 Cache Size:                                 %d bytes\n", device_l2_cache_size);
        printf(" Total amount of constant memory:               %lld bytes\n", device_total_const_mem);
        printf(" Total amount of shared memory per block:       %lld bytes\n", device_shared_mem_per_block);
        printf(" Total number of registers available per block: %d\n", device_regs_per_block);
        printf(" Warp Size:                                     %d\n", device_warp_size);
        printf(" Maximum number of threads per multiprocessor:  %d\n", device_max_threads_per_multi_processor);
        printf(" Maximum number of threads per block:           %d\n", device_max_threads_per_block);
        printf(" Max dimension size of a thread block (x,y,z):  (%d, %d, %d)\n", device_max_threads_dim[0], device_max_threads_dim[1], device_max_threads_dim[2]);
        printf(" Max dimension size of a grid size (x,y,z):     (%d, %d, %d)\n", device_max_grid_size[0], device_max_grid_size[1], device_max_grid_size[2]);
    }
    return 0;
}